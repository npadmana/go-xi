#include <stdio.h>
// CUDA-C includes
#include <hip/hip_runtime.h>


extern "C" {


void checkCUDAError(char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                             hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

void allocDev(long nbytes, void **ptr) {
	hipMalloc(ptr, nbytes);
}

void freeDev(void *ptr) {
	hipFree(ptr);
}

int sizef4() {
	return int(sizeof(float4));
}

int sizeull() {
	return int(sizeof(unsigned long long));
}

void copyToDevice(void* dst, const void* src, long nbytes) {
	hipMemcpy(dst, src, nbytes,  hipMemcpyHostToDevice);
}

void copyFromDevice(void* dst, const void* src, long nbytes) {
	hipMemcpy(dst, src, nbytes,  hipMemcpyDeviceToHost);
}

void deviceSync() {
    hipDeviceSynchronize();
}


}
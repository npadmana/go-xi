#include <stdio.h>
// CUDA-C includes
#include <hip/hip_runtime.h>


extern "C" {

const int BUFHIST=1024;


__global__ void shared_smu_kernel
(float4 *x1, int start1, int end1, 
    float4 *x2, int start2, int end2, 
    float scale, 
    int Nr, int Nmu, float invdr, unsigned long long *hist) {

    // Keep a shared copy of the histogram
    __shared__ long long _hist[BUFHIST];

    // Variable declarations
    int stride1, stride2, nh1, nr1, rstart, rend, ih;
    int ii, jj, ir, imu;
    float4 _x1, _x2;
    float s2, l2, sl, s1, l1;

    // Strides -- we will distribute over both x1 and x2
    stride1 = blockDim.y * gridDim.y;
    stride2 = blockDim.x * gridDim.x;


    // Compute the number of histograms we need to do
    nr1 = BUFHIST/Nmu;
    nh1 = (Nr + nr1 - 1)/nr1;


    // Do each piece of the histogram separately
    for (ih = 0; ih < nh1; ++ih) {
        rstart = ih*nr1;
        rend = rstart + nr1;

        // zero histogram
        // For simplicity, only a few threads will participate
        if (threadIdx.y == 0) {
            ii = threadIdx.x;
            while (ii < BUFHIST) {
                _hist[ii] = 0ll;
                ii += blockDim.x;
            }
        }
        __syncthreads();


        // Start loop over first set of data
        ii = threadIdx.y + blockIdx.y * blockDim.y + start1;
        while (ii < end1) {
            _x1 = x1[ii];
            jj = threadIdx.x + blockIdx.x * blockDim.x + start2;
            while (jj < end2) {
                _x2 = x2[jj];

                // X
                s1 = _x1.x - _x2.x;
                l1 = 0.5*(_x1.x + _x2.x);
                s2 = s1*s1;
                l2 = l1*l1;
                sl = s1*l1;

                // Y
                s1 = _x1.y - _x2.y;
                l1 = 0.5*(_x1.y + _x2.y);
                s2 += s1*s1;
                l2 += l1*l1;
                sl += s1*l1;

                // Z
                s1 = _x1.z - _x2.z;
                l1 = 0.5*(_x1.z + _x2.z);
                s2 += s1*s1;
                l2 += l1*l1;
                sl += s1*l1;

                // Compute s1, s2
                s1 = sqrtf(s2);
                l1 = rsqrtf(s2*l2 + 1.e-15);
                l1 = sl * l1;  // This is now mu, but save a register

                // Work out indices
                if (l1 < 0) {
                    l1 = -l1;
                }
                ir = s1 * invdr;
                imu = l1 * Nmu;
                if ((ir >= rstart) && (ir < rend)) {
                    atomicAdd( (unsigned long long*) &_hist[(ir-rstart)*Nmu + imu], _x1.w*_x2.w*scale);
                }

                // Loop over 2 ends
                jj += stride2;    
            }

            // Loop over 1 ends
            ii += stride1;
        }

        // Synchronize
        __syncthreads();

        // Copy histogram 
        // For simplicity, only a few threads will participate
        if (threadIdx.y == 0) {
            ir = Nmu*rstart;
            ii = threadIdx.x + ir;
            jj = Nmu*rend;
            while (ii < jj) {
                atomicAdd( (unsigned long long*) &hist[ii], _hist[ii-ir]);
                ii += blockDim.x;
            }
        }
        __syncthreads();

    
    // End histogram loop    
    }

}


int smu(void *p1, int start1, int end1, 
    void *p2, int start2, int end2, 
    float scale, 
    int Nr, int Nmu, float invdr, void *hist, 
    int nblocks, int dimx, int dimy) {

    dim3 Nb(nblocks);
    dim3 Nt(dimx, dimy);

    if (Nmu > BUFHIST) {
        return -1;
    }

    shared_smu_kernel<<<Nb, Nt>>>( (float4*) p1, start1, end1, 
    (float4*) p2, start2, end2, scale, 
    Nr, Nmu, invdr, (unsigned long long *)hist);

    return 0;
}




}